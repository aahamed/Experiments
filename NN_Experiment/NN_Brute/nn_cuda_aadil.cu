#include "hip/hip_runtime.h"
/*
 * nn_cuda_aadil.cu
 * Nearest Neighbor
 *
 */

#include <cstdio>
#include <stdio.h>
#include <cstdlib>
#include <sys/time.h>
#include <float.h>
#include <vector>
#include <sys/time.h>
#include <string>
#include <fstream>
#include <ANN/ANN.h>
#include "hip/hip_runtime.h"

//#define DEBUG
#define DELTA 1E-6
#define THREADS_PER_BLOCK 512
#define KDTREE_DIM 2
#define square(x) ((x)*(x))

using namespace std;


struct Point
{
    float coords[KDTREE_DIM];
};

typedef struct latLong
{
  float lat;
  float lng;
} LatLong;

// Host Function Prototypes
float serial_distance(Point *p, Point *q);
int serial_min_index(float *distances, int N);
void serial_nn(Point *data, Point *queries, Point *result, int N);
bool match(Point *device, Point *host, int N);
double TimeDiff(timeval t1, timeval t2);
void loadVector(const char* filename, vector<Point> &v);
void write_result(const char* filename, int N, double time);
void SearchANN(const vector <Point> &queries, const vector <Point> &data, vector <int> &idxs, vector <float> dist_sq, double &create_time, double &search_time);

// Device Function Prototypes
__device__ float distance(Point *p, Point *q);
__device__ int min_index(float* distances, int N);
__global__ void nn(Point *data, Point *queries, Point *result, int N);



// Distance Calculation (squared)
float serial_distance(Point *p, Point *q)
{
  float sdistance = 0;
  for(int i = 0; i < KDTREE_DIM; i++)
  {
    sdistance += square( p->coords[i] - q->coords[i] );
  }
  return sdistance;
}

// Find index with minimum distance
int serial_min_index(float *distances, int N)
{
  float min = distances[0];
  int min_i = 0;
  for(int i = 1; i < N; i++)
  {
    if(distances[i] < min)
    {
      min = distances[i];
      min_i = i;
    }
  }
  return min_i;
}

// Serial NN Search
void serial_nn(Point *data, Point *queries, Point *results, int N)
{
  float *distances = (float *)malloc(sizeof(float) * N);
  for(int i = 0; i < N; i++)
  {
    for(int j = 0; j < N; j++)
    {
      distances[j] = serial_distance(&queries[i], &data[j]); 
    }
    results[i] = data[serial_min_index(distances, N)];
  }
  free(distances);
}

// Checks if serial and parallel results match
bool match(Point *device, Point *host, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(serial_distance(&device[i], &host[i]) > DELTA)
    {
      printf("first mismatch at index %d\n", i);
      return false;
    }
  }
  return true;
}

void SearchANN(const vector <Point> &queries, const vector <Point> &data, vector <int> &idxs, vector <float> dist_sq, double &create_time, double &search_time)
{
    int k = 1;
    timeval t1, t2;

    idxs.resize(queries.size());
    dist_sq.resize(queries.size());

    ANNidxArray nnIdx = new ANNidx[k];
    ANNdistArray dists = new ANNdist[k];
    ANNpoint queryPt = annAllocPt(KDTREE_DIM);

    ANNpointArray dataPts = annAllocPts(data.size(), KDTREE_DIM);

    for(unsigned int i=0; i < data.size(); i++) {
        for(int j=0; j < KDTREE_DIM; j++ ) {
            dataPts[i][j] = data[i].coords[j];
        }
    }

    gettimeofday(&t1, NULL);
    ANNkd_tree* kdTree = new ANNkd_tree(dataPts, data.size(), KDTREE_DIM);
    gettimeofday(&t2, NULL);
    create_time = TimeDiff(t1,t2);

    gettimeofday(&t1, NULL);
    for(int i=0; i < queries.size(); i++) {
        for(int j=0; j < KDTREE_DIM; j++) {
            queryPt[j] = queries[i].coords[j];
        }

        kdTree->annkSearch(queryPt, 1, nnIdx, dists);

        idxs[i] = nnIdx[0];
        dist_sq[i] = dists[0];
    }
    gettimeofday(&t2, NULL);
    search_time = TimeDiff(t1,t2);

	delete [] nnIdx;
	delete [] dists;
	delete kdTree;
	annDeallocPts(dataPts);
	annClose();
}

bool verify(vector<Point> &data, vector<int> &cpu_idx, Point *gpu_res, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(serial_distance( &gpu_res[i], &data[cpu_idx[i]] ) > DELTA)
    {
      printf("first mismatch at index %s\n",i);
      return false;
    }
  }
  return true;
}

//Calculate the time difference between t1 and t2 -> outputs in ms
double TimeDiff(timeval t1, timeval t2)
{
    double t;
    t = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
    t += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms

    return t;
}

// Load data from filename into vector
void loadVector(const char* filename, vector<Point> &v)
{
  ifstream file(filename);
  char* pEnd;
  string value;
  Point l;
  if(file.is_open())
  {
    while( getline(file, value) )
    {
      l.coords[0] = strtof(value.c_str(), &pEnd);
      l.coords[1] = strtof(pEnd, NULL);
      v.push_back(l);
    }
  }
  file.close();
#ifdef DEBUG2
  for(int i = 0; i < v.size(); i++)
  {
    printf("%.2f %.2f\n", v[i].coords[0], v[i].coords[1]);
  }
#endif
}

void load_vector_random(vector<Point> &v)
{
  for(int i = 0; i < v.size(); i++)
  {
    for(int j = 0; j < KDTREE_DIM; j++)
    {
      v[i].coords[j] = 0 + 100.0*(rand() / (1.0 + RAND_MAX));
    }
  }
}

void write_result(const char* filename, int N, double time)
{
  ofstream file;
  file.open(filename, ios::out | ios::app);
  if(file.is_open())
  {
    file << N << " " << time << "\n";
  }
  file.close();
}


// Distance calculation (squared)
__device__ float distance(Point *p, Point *q)
{
  float sdistance = 0;
  for(int i = 0; i < KDTREE_DIM; i++)
  {
    sdistance += square( p->coords[i] - q->coords[i] );
  }
  return sdistance;
}

// Find index with minimum distance
__device__ int min_index(float* distances, int N)
{
  float min = distances[0]; //inefficient ?
  int min_i = 0;
  for(int i = 1; i < N; i++)
  {
    if(distances[i] < min)
    {
      min = distances[i];
      min_i = i;
    }
  }
#ifdef DEBUG2
  if(threadIdx.x == 0)
  {
    printf("Min index for thread 0 = %d  value = %.2f\n", min_i, min);
  }
#endif
  return min_i;
}


/**
* Kernel - deprecated
* Executed on GPU
* Perform nearest neighbor search against data using queries
*/
__global__ void nn_dep(Point *data, Point *queries, Point *result, int N)
{
  int global_id = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;
  if (global_id < N)
  {
    Point* qpoint = &queries[global_id];
    float* distances = (float*)malloc(sizeof(float) * N);
    for(int i = 0; i < N; i++)
    {
      distances[i] = distance(qpoint, &data[i]);
    }
    result[global_id] = data[min_index(distances, N)];
    free(distances);
  }

}


/**
* Kernel
* Executed on GPU
* Perform nearest neighbor search against data using queries
*/
__global__ void nn(Point *data, Point *queries, Point *result, int N)
{
  int global_id = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;
  if (global_id < N)
  {
    Point* qpoint = &queries[global_id];
    int min_i = 0;
    float min_distance = distance(qpoint, &data[0]);
    float t_distance = 0;
    for(int i = 1; i < N; i++)
    {
      t_distance = distance(qpoint, &data[i]);
      if(t_distance < min_distance)
      {
        min_distance = t_distance;
        min_i = i;
      }
    }
    result[global_id] = data[min_i];
  }

}

// Main body
int main()
{
  vector <Point> data;
  vector <Point> queries;
  Point *d_data, *d_queries, *d_results, *results;
  //Point *serial_results;
  timeval t1, t2;
  double elapsed_gpu;
  //double elapsed_serial;
 
  const char *data_file = "data/data.txt";
  const char *query_file = "data/query.txt";
  const char *out_file = "output/brute_out.txt";
  loadVector(data_file, data);
  loadVector(query_file, queries);
  if(queries.size() != data.size())
  {
    printf("size of query set does not match size of data set\n");
    return 1;
  }

  int N = queries.size();
  int blocks = N/THREADS_PER_BLOCK + ((N % THREADS_PER_BLOCK) ? 1 : 0);
  int size = N * sizeof(Point);

  /* initialize random seed: */
  srand (time(NULL));

  
  //Allocate space for result
  results = (Point *)malloc(size);
  //serial_results = (Point *)malloc(size);

  // Allocate space for device copies of data and queries
  hipMalloc((void **)&d_data, size);
  hipMalloc((void **)&d_queries, size);
  hipMalloc((void **)&d_results, size);

  // Copy inputs to device
  hipMemcpy(d_data, &data[0], size, hipMemcpyHostToDevice);
  hipMemcpy(d_queries, &queries[0], size, hipMemcpyHostToDevice);

  gettimeofday(&t1, NULL);
  //Launch kernel for GPU NN Search
  nn<<<blocks, THREADS_PER_BLOCK>>>(d_data, d_queries, d_results, N);
  if(hipGetLastError() != hipSuccess)
  {
    printf("%s\n", hipGetErrorString((hipGetLastError())));
  }
  gettimeofday(&t2, NULL);
  elapsed_gpu = TimeDiff(t1, t2); 
  // Copy result back to host
  hipMemcpy(results, d_results, size, hipMemcpyDeviceToHost);

  //Perform CPU NN Search
  vector<int> idx;
  vector<float> dist;
  double cpu_create_time, cpu_search_time;
  SearchANN(queries, data, idx, dist, cpu_create_time, cpu_search_time);

#ifdef DEBUG
  for(int i = 0; i < 5; i++)
  {
        printf("query: (%.2f, %.2f) gpu_nn: (%.2f, %.2f) ann_nn: (%.2f, %.2f)\n", 
            queries[i].coords[0], queries[i].coords[1], results[i].coords[0], results[i].coords[1], 
            data[idx[i]].coords[0], data[idx[i]].coords[1]);
  }
#endif

  if(verify(data, idx, results, N))
  {
    printf("Host and Device Results match !\n");
    printf("GPU Running Time: %.3f\n", elapsed_gpu);
    printf("CPU Running Time: %.3f\n", cpu_search_time);
    printf("Speedup on GPU: %.2f\n", cpu_search_time/elapsed_gpu);
    write_result(out_file, N, elapsed_gpu);
  }
  else
  {
    printf("!!!!! Host and Device Results DO NOT match !!!!!!\n");
  }

  //Cleanup
  free(results);
  hipFree(d_data);
  hipFree(d_queries);
  hipFree(d_results);
  return 0;
}


void tmp()
{

#ifdef DEBUG2
  //Do Serial NN Search
  gettimeofday(&t1, NULL);
  serial_nn(&data[0], &queries[0], serial_results, N);
  gettimeofday(&t2, NULL);
  elapsed_serial = TimeDiff(t1, t2);

  for(int i = 0; i < 5; i++)
  {
    printf("query: (%.2f, %.2f) gpu_nn: (%.2f, %.2f) ser_nn(%.2f, %.2f)\n", 
        queries[i].coords[0], queries[i].coords[1], results[i].coords[0], results[i].coords[1], 
        serial_results[i].coords[0], serial_results[i].coords[1]);
  }
  
  if(match(results, serial_results, N))
  {
    printf("Host and Device Results match !\n");
    printf("GPU Running Time: %.3f\n", elapsed_gpu);
    printf("Serial Running Time: %.3f\n", elapsed_serial);
    printf("Speedup on GPU: %.2f\n", elapsed_serial/elapsed_gpu);
    write_result(out_file, N, elapsed_gpu);
  }
  else
  {
    printf("Host and Device Results DO NOT match !\n");
  }

#endif
}
